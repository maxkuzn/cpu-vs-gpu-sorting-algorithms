#include "hip/hip_runtime.h"
#include "merge_sort.h"

#include "hip/hip_runtime.h"
#include ""

#include <vector>



namespace gpu {
  namespace {

  template <typename T>
  void bottom_up_merge_chunks_impl(std::vector<T>& data,
                                   std::vector<T>& buffer,
                                   size_t chunk_size,
                                   size_t chunk_pair_idx) {
      
    size_t left_chunk = 2 * chunk_pair_idx;
    size_t right_chunk = 2 * chunk_pair_idx + 1;
    size_t left_idx = left_chunk * chunk_size;
    size_t left_end = left_idx + chunk_size;
    size_t right_idx = right_chunk * chunk_size;
    size_t right_end = right_idx + chunk_size;
    if (right_end > data.size()) {
      right_end = data.size();
    }
    size_t write_idx = left_chunk * chunk_size;

    while (left_idx < left_end && right_idx < right_end) {
      if (data[left_idx] < data[right_idx]) {
        buffer[write_idx++] = data[left_idx++];
      } else {
        buffer[write_idx++] = data[right_idx++];
      }
    }
    while (left_idx < left_end) {
      buffer[write_idx++] = data[left_idx++];
    }
    while (right_idx < right_end) {
      buffer[write_idx++] = data[right_idx++];
    }
  }

  template <typename T>
  void bottom_up_merge_impl(std::vector<T>& data,
                            std::vector<T>& buffer, size_t chunk_size) {
    size_t unmerged_chunks = (data.size() / chunk_size) +
                             (data.size() % chunk_size != 0);
    size_t chunk_pair_idx = 0;
    while (2 * chunk_pair_idx + 1 < unmerged_chunks) {
      bottom_up_merge_chunks_impl(data, buffer, chunk_size, chunk_pair_idx);
      ++chunk_pair_idx;
    }
  }

  template<typename T>
  __global__
  void cu_bottom_up_merge_impl(T* data, T* buffer, size_t size, size_t chunk_size) {
    // Global thread id.
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    size_t unmerged_chunks = (size / chunk_size) + (size % chunk_size != 0);
    size_t thread_left_chunk = 2 * thread_id;
    size_t thread_right_chunk = 2 * thread_id + 1;
    if (thread_right_chunk >= unmerged_chunks) {
      // Nothing to merge
      return;
    }
    size_t left_idx = thread_left_chunk * chunk_size;
    size_t left_end = left_idx + chunk_size;
    size_t right_idx = thread_right_chunk * chunk_size;
    size_t right_end = right_idx + chunk_size;
    if (right_end > size) {
      right_end = size;
    }

    size_t write_idx = thread_left_chunk * chunk_size;

    while (left_idx < left_end && right_idx < right_end) {
      if (data[left_idx] < data[right_idx]) {
        buffer[write_idx++] = data[left_idx++];
      } else {
        buffer[write_idx++] = data[right_idx++];
      }
    }
    while (left_idx < left_end) {
      buffer[write_idx++] = data[left_idx++];
    }
    while (right_idx < right_end) {
      buffer[write_idx++] = data[right_idx++];
    }
  }

  }  // void namespace

  template <typename T>
  void merge_sort(std::vector<T>& data) {
    constexpr size_t block_size = 1024;
    constexpr size_t merge_chunks_until_n_chunks = 64;

    T* cu_data;
    hipMalloc(&cu_data, data.size() * sizeof(T));
    hipMemcpy(cu_data, data.data(), data.size() * sizeof(T), hipMemcpyHostToDevice);
    T* cu_buffer;
    hipMalloc(&cu_buffer, data.size() * sizeof(T));


    size_t unmerged_chunks = data.size();
    size_t chunk_size = 1;
    while (unmerged_chunks > merge_chunks_until_n_chunks) {
      size_t total_thread_blocks = (unmerged_chunks / block_size) +
                                   (unmerged_chunks % block_size != 0);

      cu_bottom_up_merge_impl<T><<< total_thread_blocks, block_size >>>(
          cu_data, cu_buffer, data.size(), chunk_size);
      hipDeviceSynchronize();
      std::swap(cu_data, cu_buffer);

      chunk_size *= 2;
      unmerged_chunks = unmerged_chunks / 2 + (unmerged_chunks % 2 != 0);
    }
    hipMemcpy(data.data(), cu_data, data.size() * sizeof(T), hipMemcpyDeviceToHost);
    hipFree(cu_buffer);
    hipFree(cu_data);

    std::vector<T> buffer(data.size());
    while (unmerged_chunks > 1) {
      bottom_up_merge_impl<T>(data, buffer, chunk_size);
      std::swap(data, buffer);

      chunk_size *= 2;
      unmerged_chunks = unmerged_chunks / 2 + (unmerged_chunks % 2 != 0);
    }
  }

  template void merge_sort<int32_t>(std::vector<int32_t>&);
  template void merge_sort<int64_t>(std::vector<int64_t>&);
  template void merge_sort<float>(std::vector<float>&);
  template void merge_sort<double>(std::vector<double>&);
  // merge_sort<int32_t>;
  // merge_sort<int64_t>;
  // merge_sort<float>;
  // merge_sort<double>;
}
